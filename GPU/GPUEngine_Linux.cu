#include "hip/hip_runtime.h"
/*
 * This file is part of the VanitySearch distribution (https://github.com/JeanLucPons/VanitySearch).
 * Copyright (c) 2019 Jean Luc PONS.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef WIN64
#include <unistd.h>
#endif
#include <stdio.h>

#include "GPUEngine.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdint.h>
#include "../hash/sha256.h"
#include "../hash/ripemd160.h"
#include "../Timer.h"

#include "GPUGroup.h"
#include "GPUMath.h"
#include "GPUHash.h"
#include "GPUBase58.h"
#include "GPUWildcard.h"
#include "GPUCompute.h"

// ---------------------------------------------------------------------------------------

__global__ void comp_keys(uint32_t mode,prefix_t *prefix, uint32_t *lookup32, uint64_t *keys, uint32_t maxFound, uint32_t *found) {

  int xPtr = (blockIdx.x*blockDim.x) * 8;
  int yPtr = xPtr + 4 * blockDim.x;
  ComputeKeys(mode, keys + xPtr, keys + yPtr, prefix, lookup32, maxFound, found);

}

__global__ void comp_keys_p2sh(uint32_t mode, prefix_t *prefix, uint32_t *lookup32, uint64_t *keys, uint32_t maxFound, uint32_t *found) {

  int xPtr = (blockIdx.x*blockDim.x) * 8;
  int yPtr = xPtr + 4 * blockDim.x;
  ComputeKeysP2SH(mode, keys + xPtr, keys + yPtr, prefix, lookup32, maxFound, found);

}

__global__ void comp_keys_comp(prefix_t *prefix, uint32_t *lookup32, uint64_t *keys, uint32_t maxFound, uint32_t *found) {

  int xPtr = (blockIdx.x*blockDim.x) * 8;
  int yPtr = xPtr + 4 * blockDim.x;
  ComputeKeysComp(keys + xPtr, keys + yPtr, prefix, lookup32, maxFound, found);

}

__global__ void comp_keys_pattern(uint32_t mode, prefix_t *pattern, uint64_t *keys,  uint32_t maxFound, uint32_t *found) {

  int xPtr = (blockIdx.x*blockDim.x) * 8;
  int yPtr = xPtr + 4 * blockDim.x;
  ComputeKeys(mode, keys + xPtr, keys + yPtr, NULL, (uint32_t *)pattern, maxFound, found);

}

__global__ void comp_keys_p2sh_pattern(uint32_t mode, prefix_t *pattern, uint64_t *keys, uint32_t maxFound, uint32_t *found) {

  int xPtr = (blockIdx.x*blockDim.x) * 8;
  int yPtr = xPtr + 4 * blockDim.x;
  ComputeKeysP2SH(mode, keys + xPtr, keys + yPtr, NULL, (uint32_t *)pattern, maxFound, found);

}

//#define FULLCHECK
#ifdef FULLCHECK

// ---------------------------------------------------------------------------------------

__global__ void chekc_mult(uint64_t *a, uint64_t *b, uint64_t *r) {

  _ModMult(r, a, b);
  r[4]=0;

}

// ---------------------------------------------------------------------------------------

__global__ void chekc_hash160(uint64_t *x, uint64_t *y, uint32_t *h) {

  _GetHash160(x, y, (uint8_t *)h);
  _GetHash160Comp(x, y, (uint8_t *)(h+5));

}

// ---------------------------------------------------------------------------------------

__global__ void get_endianness(uint32_t *endian) {

  uint32_t a = 0x01020304;
  uint8_t fb = *(uint8_t *)(&a);
  *endian = (fb==0x04);

}

#endif //FULLCHECK

// ---------------------------------------------------------------------------------------

using namespace std;

std::string toHex(unsigned char *data, int length) {

  string ret;
  char tmp[3];
  for (int i = 0; i < length; i++) {
    if (i && i % 4 == 0) ret.append(" ");
    sprintf(tmp, "%02x", (int)data[i]);
    ret.append(tmp);
  }
  return ret;

}

int _ConvertSMVer2Cores(int major, int minor) {

  // Defines for GPU Architecture types (using the SM version to determine
  // the # of cores per SM
  typedef struct {
    int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
    // and m = SM minor version
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = {
      {0x20, 32}, // Fermi Generation (SM 2.0) GF100 class
      {0x21, 48}, // Fermi Generation (SM 2.1) GF10x class
      {0x30, 192},
      {0x32, 192},
      {0x35, 192},
      {0x37, 192},
      {0x50, 128},
      {0x52, 128},
      {0x53, 128},
      {0x60,  64},
      {0x61, 128},
      {0x62, 128},
      {0x70,  64},
      {0x72,  64},
      {0x75,  64},
      {0x80,  64},
      {0x86, 128},
      {-1, -1} };

  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1) {
    if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchCoresPerSM[index].Cores;
    }

    index++;
  }

  return 0;

}

GPUEngine::GPUEngine(int nbThreadGroup, int nbThreadPerGroup, int gpuId, uint32_t maxFound,bool rekey) {

  // Initialise CUDA
  this->rekey = rekey;
  this->nbThreadPerGroup = nbThreadPerGroup;
  initialised = false;
  hipError_t err;

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess) {
    printf("GPUEngine: CudaGetDeviceCount %s %d\n", hipGetErrorString(error_id),error_id);
    return;
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0) {
    printf("GPUEngine: There are no available device(s) that support CUDA\n");
    return;
  }

  err = hipSetDevice(gpuId);
  if (err != hipSuccess) {
    printf("GPUEngine: %s\n", hipGetErrorString(err));
    return;
  }

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, gpuId);

  if (nbThreadGroup == -1)
    nbThreadGroup = deviceProp.multiProcessorCount * 8;

  this->nbThread = nbThreadGroup * nbThreadPerGroup;
  this->maxFound = maxFound;
  this->outputSize = (maxFound*ITEM_SIZE + 4);

  char tmp[512];
  sprintf(tmp,"GPU #%d %s (%dx%d cores) Grid(%dx%d)",
  gpuId,deviceProp.name,deviceProp.multiProcessorCount,
  _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
                      nbThread / nbThreadPerGroup,
                      nbThreadPerGroup);
  deviceName = std::string(tmp);

  // Prefer L1 (We do not use __shared__ at all)
  err = hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  if (err != hipSuccess) {
    printf("GPUEngine: %s\n", hipGetErrorString(err));
    return;
  }

  size_t stackSize = 49152;
  err = hipDeviceSetLimit(hipLimitStackSize, stackSize);
  if (err != hipSuccess) {
    printf("GPUEngine: %s\n", hipGetErrorString(err));
    return;
  }

  // rest of the code ...
} 